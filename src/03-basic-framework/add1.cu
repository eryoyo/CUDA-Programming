// *****************头文件包含*****************

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

// *****************常量定义*****************
const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

// *****************函数声明*****************
void __global__ add(const double *x, const double *y, double *z);
void check(const double *z, const int N);

int main(void)
{
    // *****************分配主机内存*****************
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    // *****************分配设备内存*****************
    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);

    // *****************将某些数据从主机复制到设备*****************
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    // *****************调用核函数*****************
    const int block_size = 128;
    const int grid_size = N / block_size;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z);

    // *****************将某些数据从设备复制到主机*****************
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    // *****************释放主机与设备内存*****************
    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;
}

void __global__ add(const double *x, const double *y, double *z)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    z[n] = x[n] + y[n];
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

