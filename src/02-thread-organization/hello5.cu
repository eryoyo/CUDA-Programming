
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    // gridDim, blockDim都是dim3类型的变量，有x, y, z三个变量
    // blockIdx, threadIdx是uint3类型的变量，也有x, y, z三个变量
    // 多维的网格和线程块本质上还是1维的，一个多维线程指标threadIdx.x、threadIdx.y、threadIdx.z 对应的一维指标为
    // int tid = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x
    // 与一个多维线程块指标blockIdx.x、blockIdx.y、blockIdx.z 对应的一维指标没有唯一的定义（主要是因为各个线程块的执行是相互独立的）
    const int b = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    printf("Hello World from block-%d and thread-(%d, %d)!\n", b, tx, ty);
}

int main(void)
{
    // 一个线程块中的线程还可以细分为不同的线程束（thread warp）。一个线程束（即一束线程）是同一个线程块中相邻的warpSize个线程。
    // warpSize 也是一个内建变量，表示线程束大小，其值对于目前所有的GPU 架构都是32。所以，一个线程束就是连续的32个线程。具体地说，
    // 一个线程块中第0到第31个线程属于第0个线程束，第32到第63个线程属于第1个线程束，依此类推。
    const dim3 block_size(2, 4);
    hello_from_gpu<<<1, block_size>>>();
    hipDeviceSynchronize();
    return 0;
}

