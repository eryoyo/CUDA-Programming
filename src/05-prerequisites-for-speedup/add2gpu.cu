#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h>
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
    const real EPSILON = 1.0e-15;
#else
    typedef float real;
    const real EPSILON = 1.0e-6f;
#endif

const int NUM_REPEATS = 10;
const real a = 1.23;
const real b = 2.34;
const real c = 3.57;
void __global__ add(const real *x, const real *y, real *z, const int N);
void check(const real *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(real) * N;
    real *h_x = (real*) malloc(M);
    real *h_y = (real*) malloc(M);
    real *h_z = (real*) malloc(M);

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    real *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void **)&d_x, M));
    CHECK(hipMalloc((void **)&d_y, M));
    CHECK(hipMalloc((void **)&d_z, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        // cuda事件类型
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        // 对于处于TCC驱动模式的GPU可以省略，但是对于WDDM驱动模式下的GPU必须保留
        // 因为WDDM模式下一个CUDA stream中的操作，例如此处的hipEventRecord函数不是直接交给GPU执行，而是先提交到一个软件队列
        // 需要添加一条对该流的 hipEventQuery 操作（或者hipEventSynchronize）刷新队列，才能促使前面的操作在GPU执行
        hipEventQuery(start);

        add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        // 计算两个事件之间的时间差
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));
    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));
    return 0;
}

void __global__ add(const real *x, const real *y, real *z, const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
}

void check(const real *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

